#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include "cuda_ranges_to_cloud_lib.cuh"

#include <gflags/gflags.h>

namespace laser_processing
{

  __global__ void convertRangeToPoint(int num_points, double angle_min, double angle_inc, double range_min, double range_max, 
    float laser_loc_x, float laser_loc_y, float *ranges, float *points_x, float *points_y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < num_points; i += stride) {
      const float a = angle_min + static_cast<float>(i) * angle_inc;
      
      const float r = ((ranges[i] > range_min && ranges[i] < range_max) ? ranges[i] : range_max);
      points_x[i] = r * cos(a) + laser_loc_x;
      points_y[i] = r * sin(a) + laser_loc_y;
    }
  }

  std::vector<Vector2f> LaserCallback(const sensor_msgs::LaserScan &msg) {

    // Location of the laser on the robot. Assumes the laser is forward-facing.
    const Vector2f kLaserLoc(0.2, 0);

    std::vector<Vector2f> point_cloud;

    int num_ranges = msg.ranges.size();
    int num_bytes = num_ranges * sizeof(float);
    float *points_x_device;
    float *points_y_device;
    float *ranges_device;

    float points_x_host[num_ranges];
    float points_y_host[num_ranges];
    float ranges_host[num_ranges];

    copy(msg.ranges.begin(),msg.ranges.end(),ranges_host);

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMalloc(&points_x_device, num_bytes);
    hipMalloc(&points_y_device, num_bytes);
    hipMalloc(&ranges_device, num_bytes);

    hipMemcpy(ranges_device, ranges_host, num_bytes, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (num_ranges + blockSize - 1) / blockSize;
    convertRangeToPoint<<<numBlocks, blockSize>>>(
      num_ranges, msg.angle_min, msg.angle_increment, msg.range_min, msg.range_max, 
      kLaserLoc.x_, kLaserLoc.y_, ranges_device, points_x_device, points_y_device);

    hipMemcpy(points_x_host, points_x_device, num_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(points_y_host, points_y_device, num_bytes, hipMemcpyDeviceToHost);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Free memory
    hipFree(points_x_device);
    hipFree(points_y_device);
    hipFree(ranges_device);

    for (size_t point_idx = 0; point_idx < num_ranges; point_idx++) {
      point_cloud.emplace_back(Vector2f(points_x_host[point_idx], points_y_host[point_idx]));
    }

    return point_cloud;
  }

}